#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

extern "C"
{
#include "image.h"
#include "stencil.h"
}

#define TILE_SIZE 16
#define BLOCK_SIZE 16

__constant__ float  G_St[3][3];

__global__ void stencil_kernel (int *h, int *w, float *St,
        float *InR, float *InG, float *InB,
        float *OutR, float *OutG, float *OutB)
{
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int x=blockIdx.x*BLOCK_SIZE+tx;
    int y=blockIdx.y*BLOCK_SIZE+ty;

#if 0    // Simple implementation with Constant memory - Working like a Don!!
    if(x<*w && y<*h) {
        float R=0.,G=0.,B=0.;
        for(int i=0; i<3; ++i)
        {
            for(int j=0; j<3; ++j)
            {
                int const xpos = x+(i-1);
                int const ypos = y+(j-1);
                //float const x = *(St+3*i+j);
                float const x = G_St[i][j];
                R += *(InR+(ypos**w+xpos)) * x;
                G += *(InG+(ypos**w+xpos)) * x;
                B += *(InB+(ypos**w+xpos)) * x;
            }
        }
        *(OutR+(y**w+x)) = R;
        *(OutG+(y**w+x)) = G;
        *(OutB+(y**w+x)) = B;
    }
#elif 1
    __shared__ float sR[BLOCK_SIZE+2][BLOCK_SIZE+2];
    __shared__ float sG[BLOCK_SIZE+2][BLOCK_SIZE+2];
    __shared__ float sB[BLOCK_SIZE+2][BLOCK_SIZE+2];

    // Adjust the co-ordinates
    x -=1;    y -=1;
    // Put Input data in shared memory
    if(x>=0 && x<*w && y>=0 && y<*h) {
        sR[tx][ty]=*(InR+(y**w+x));
        sG[tx][ty]=*(InR+(y**w+x));
        sB[tx][ty]=*(InR+(y**w+x));
    } else {
        sR[tx][ty] = 0.;
        sG[tx][ty] = 0.;
        sB[tx][ty] = 0.;
    }
    __syncthreads();

    if(tx<BLOCK_SIZE && ty<BLOCK_SIZE) {
        float R=0.,G=0.,B=0.;
        for(int i=0; i<3; ++i)
            for(int j=0; j<3; ++j)
            {
                float const x = G_St[i][j];
                R += sR[tx+(i-1)][ty+(j-1)] * x;
                G += sG[tx+(i-1)][ty+(j-1)] * x;
                B += sB[tx+(i-1)][ty+(j-1)] * x;
            }
        __syncthreads();
        if(x<*w && y<*h) {
        *(OutR+(y**w+x)) = R;
        *(OutG+(y**w+x)) = G;
        *(OutB+(y**w+x)) = B;
      }
    }
#elif 0
    // Put Stencil in constant memory
        float R=0.,G=0.,B=0.;
        for(int i=0; i<3; ++i)
            for(int j=0; j<3; ++j)
            {
                int xpos = x+(i-1);
                int ypos = y+(j-1);
                R += *(InR+(ypos**w+xpos)) * G_St[i][j];
                G += *(InG+(ypos**w+xpos)) * G_St[i][j];
                B += *(InB+(ypos**w+xpos)) * G_St[i][j];
            }
        *(OutR+(y**w+x)) = R;
        *(OutG+(y**w+x)) = G;
        *(OutB+(y**w+x)) = B;
#elif 0
    // Process a bunch of data points in one thread
#endif
}

image_t * stencil_cuda(
    image_t const * const input,
    float stencil[3][3],
    int const num_times)
{
  float *Ired, *Iblue, *Igreen;
  float *Ored, *Oblue, *Ogreen;
  int *d_h, *d_w;
  float *d_stencil;

  const int h=input->height;
  const int w=input->width;
  hipError_t rv;

  // Allocate memory on the device
  if(hipSuccess != (rv = hipMalloc((void **)&d_h, sizeof(int))))              printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));
  if(hipSuccess != (rv = hipMalloc((void **)&d_w, sizeof(int))))              printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));
  if(hipSuccess != (rv = hipMalloc((void **)&d_stencil, sizeof(float)*3*3)))  printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));

  if(hipSuccess != (rv = hipMalloc((void **)&Ired,   h*w*sizeof(float))))      printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));
  if(hipSuccess != (rv = hipMalloc((void **)&Igreen, h*w*sizeof(float))))      printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));
  if(hipSuccess != (rv = hipMalloc((void **)&Iblue,  h*w*sizeof(float))))      printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));

  if(hipSuccess != (rv = hipMalloc((void **)&Ored,   h*w*sizeof(float))))      printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));
  if(hipSuccess != (rv = hipMalloc((void **)&Ogreen, h*w*sizeof(float))))      printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));
  if(hipSuccess != (rv = hipMalloc((void **)&Oblue,  h*w*sizeof(float))))      printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));

  // Copy data to GPU memory
  // Just copy the width and height
  hipMemcpy(d_h, &input->height, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_w, &input->width,  sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(Ired,   input->red,   h*w*sizeof(*input->red),   hipMemcpyHostToDevice);
  hipMemcpy(Igreen, input->green, h*w*sizeof(*input->green), hipMemcpyHostToDevice);
  hipMemcpy(Iblue,  input->blue,  h*w*sizeof(*input->blue),  hipMemcpyHostToDevice);

  if(hipSuccess != (rv=hipMemcpy(d_stencil, &stencil,  sizeof(float)*3*3,  hipMemcpyHostToDevice)))     printf("rv = %d = %s.\n", rv, hipGetErrorString(rv));

  if(hipSuccess != (rv=hipMemcpyToSymbol(HIP_SYMBOL(G_St), stencil, 3*3*sizeof(float))))
      printf("Const mem error! sizeof(%lu): Err msg %s  \n", 3*3*sizeof(float), hipGetErrorString(rv));

  hipDeviceSynchronize();

  dim3 dimBlock(BLOCK_SIZE+2, BLOCK_SIZE+2,1);
  dim3 dimGrid(ceil(float (w)/BLOCK_SIZE), ceil(float (h)/BLOCK_SIZE),1);

  printf("Image (w=%d x h=%d).\n", w, h);
  printf("Grids %dx%d Blocks %dx%d. St[1][1] %f\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y, stencil[1][1]);

  for(int i=0; i<num_times ;++i)
  {
      printf("Calling kernel .... \n");
      stencil_kernel<<<dimGrid, dimBlock>>>(d_h, d_w, d_stencil,
              Ired, Igreen, Iblue,
              Ored, Ogreen, Oblue);
      if ( hipSuccess != (rv=hipGetLastError()) )
          printf( "**** Error! %d: %s ****\n", rv, hipGetErrorString(rv) );
  }

  image_t * output = image_alloc(input->width, input->height);

  // Get the result from GPU
  hipMemcpy(output->red,   Ored,   h*w*sizeof(*input->red),   hipMemcpyDeviceToHost);
  hipMemcpy(output->green, Ogreen, h*w*sizeof(*input->green), hipMemcpyDeviceToHost);
  hipMemcpy(output->blue,  Oblue,  h*w*sizeof(*input->blue),  hipMemcpyDeviceToHost);

  hipFree(d_w); hipFree(d_h); hipFree(d_stencil);
  hipFree(Ired); hipFree(Iblue); hipFree(Igreen);
  hipFree(Ored); hipFree(Oblue); hipFree(Ogreen);
  return output;
}
